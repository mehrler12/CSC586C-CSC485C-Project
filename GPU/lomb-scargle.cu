
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>

int const blocksize = 512;


__global__
void periodogram_frequency( double *time, double *flux, double *frequency, double *periodogram, double one_over_2variance, size_t sample_size, size_t num_frequencies){
    // Calculate frequency index

    //One thing, printf works from the kernel but only up to a certain length then it just doesn't print anymore. Learnt that the hard way...
    int const w = threadIdx.x + blockIdx.x * blockDim.x;
    if ( w < num_frequencies) { 
       
        double cos_sum_squared = 0llu, cos_squared_sum = 0llu, sin_sum_squared = 0llu, sin_squared_sum = 0llu;
       
        
        double frequency_2 = 2 * frequency[w];
   
        //A potential way forward might be to split the frequencies across blocks and this across threads? I had to remove the temp arrays as there wasn't enough memory
        //Also not really sure we need them anymore as they were mainly for vectorization
        for(int i = 0; i< sample_size;i++){
            double coscal;
            double sincal;
            sincospi(frequency_2 * time[i],&sincal,&coscal);
            cos_sum_squared = fma(flux[i],coscal,cos_sum_squared);
            cos_squared_sum = fma(coscal, coscal,cos_squared_sum);
            sin_sum_squared = fma(flux[i], sincal,sin_sum_squared);
            sin_squared_sum = fma(sincal, sincal,sin_squared_sum);
        }
        cos_sum_squared = cos_sum_squared * cos_sum_squared;
        sin_sum_squared = sin_sum_squared * sin_sum_squared;

        periodogram[w] = one_over_2variance * (cos_sum_squared/cos_squared_sum + sin_sum_squared/sin_squared_sum);
    }
}

void lomb_scargle(double *flux, double *time, double *frequency, double *periodogram, double variance, size_t sample_size, size_t num_frequencies){

    double *time2 = new double[sample_size];
    double *flux2 = new double[sample_size];
    
    double flux_total = 0llu;
    for(int i = 0; i<sample_size ; i++){
        flux_total =  flux_total + flux[i];
    }

    double flux_avg = flux_total/sample_size;
    
    double initial_time = time[0];
    for(int i = 0 ; i < sample_size ; i++){
        time2[i] = time[i] - initial_time;
        flux2[i] = flux[i] - flux_avg;
    }

    double *dev_time, *dev_flux, *dev_frequency, *dev_periodogram;

    hipMalloc( (void **) &dev_time, sample_size * sizeof(double));
    hipMalloc( (void **) &dev_flux, sample_size * sizeof(double));
    hipMalloc( (void **) &dev_frequency, num_frequencies * sizeof(double));
    hipMalloc( (void **) &dev_periodogram, num_frequencies * sizeof(double));


    hipMemcpy( dev_time, time2, sample_size * sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy( dev_flux, flux2, sample_size * sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy( dev_frequency, frequency, num_frequencies * sizeof(double), hipMemcpyHostToDevice );

    double one_over_2variance = 1 / (2 * variance);

    auto const num_blocks = ceil(num_frequencies/ static_cast< float >(blocksize));

    periodogram_frequency<<< num_blocks, blocksize >>>(dev_time, dev_flux, dev_frequency, dev_periodogram, one_over_2variance, sample_size, num_frequencies);
    
    //Cuda fails silently, need this to see errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy( periodogram, dev_periodogram, num_frequencies * sizeof(double), hipMemcpyDeviceToHost );
}

