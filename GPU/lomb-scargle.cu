
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>

int const blocksize = 512;

__global__
void periodogram_frequency(double *flux, double *time, double *frequency, double *periodogram, double one_over_2variance, size_t sample_size, size_t num_frequencies){
    // Calculate frequency index
    int const w = threadIdx.x + blockIdx.x * blockDim.x;

    if ( w < num_frequencies) {    
        double cos_sum_squared = 0llu, cos_squared_sum = 0llu, sin_sum_squared = 0llu, sin_squared_sum = 0llu;
        // This gave an error "error: expression must have a constant value"
        // I think we might need to allocate all memory of dynamic size on the host and pass it in - not sure though, I need to look into this more
        // We could allocate arrays of size sample_size * num_frequencies and pass them in
        double time2[sample_size], sins[sample_size], coss[sample_size];
        double frequency_2_pi = 2 * M_PI * frequency[w];

        for(int i = 0; i < sample_size ; i++){
            time2[i] = frequency_2_pi * time[i];
        }
        for (int i = 0 ; i < sample_size ; i++){
            coss[i] = cos(time2[i]);
        }
        for (int i = 0 ; i < sample_size ; i++){
            sins[i] = sin(time2[i]);
        }
        for(int i = 0; i< sample_size;i++){
            cos_sum_squared += flux[i] * coss[i];
            cos_squared_sum += coss[i] * coss[i];
            sin_sum_squared += flux[i] * sins[i];
            sin_squared_sum += sins[i] * sins[i];
        }
        cos_sum_squared = cos_sum_squared * cos_sum_squared;
        sin_sum_squared = sin_sum_squared * sin_sum_squared;

        periodogram[w] = one_over_2variance * (cos_sum_squared/cos_squared_sum + sin_sum_squared/sin_squared_sum);
    }
}

void lomb_scargle(double *flux, double *time, double *frequency, double *periodogram, double variance){

    int sample_size = sizeof(flux)/sizeof(flux[0]);
    int num_frequencies = sizeof(frequency)/sizeof(frequency[0]);

    double flux_total = 0;
    for(int i = 0; i<sample_size ; i++){
        flux_total += flux[i];
    }
    double flux_avg = flux_total/sample_size;
    
    double initial_time = time[0];
    for(int i = 0 ; i < sample_size ; i++){
        time[i] -= initial_time;
    }

    for(int i = 0 ; i< sample_size ; i++){
        flux[i] -= flux_avg;
    }

    // Allocate memory on GPU
    double *dev_time, *dev_flux, *dev_frequency, *dev_periodogram;

    hipMalloc( (void **) &dev_time, sample_size);
    hipMalloc( (void **) &dev_flux, sample_size);
    hipMalloc( (void **) &dev_frequency, num_frequencies);
    hipMalloc( (void **) &dev_periodogram, num_frequencies);


    hipMemcpy( dev_time, time, sample_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_flux, flux, sample_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_frequency, frequency, sample_size, hipMemcpyHostToDevice );

    double one_over_2variance = 1 / (2 * variance);

    auto const num_blocks = ceil(num_frequencies/ static_cast< float >(blocksize));

    periodogram_frequency<<< num_blocks, blocksize >>>(dev_time, dev_flux, dev_frequency, dev_periodogram, one_over_2variance, sample_size, num_frequencies);

    hipMemcpy( periodogram, dev_periodogram, sample_size, hipMemcpyDeviceToHost );
}

