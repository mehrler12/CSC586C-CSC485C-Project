#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

int const blocksize = 512;

void lomb_scargle(double *flux, double *time, double *frequency, double *periodogram, double variance){

    int sample_size = sizeof(flux)/sizeof(flux[0]);
    int num_frequencies = sizeof(frequency)/sizeof(frequency[0]);

    double flux_total = 0;
    for(int i = 0; i<sample_size ; i++){
        flux_total += flux[i];
    }
    double flux_avg = flux_total/sample_size;
    
    double initial_time = time[0];
    for(int i = 0 ; i < sample_size ; i++){
        time[i] -= initial_time;
    }

    for(int i = 0 ; i< sample_size ; i++){
        flux[i] -= flux_avg;
    }

    // Allocate memory on GPU
    double *dev_time, *dev_flux, *dev_frequency, *dev_periodogram;

    hipMalloc( (void **) &dev_time, sample_size);
    hipMalloc( (void **) &dev_flux, sample_size);
    hipMalloc( (void **) &dev_frequency, num_frequencies);
    hipMalloc( (void **) &dev_periodogram, num_frequencies);


    hipMemcpy( dev_time, time, sample_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_flux, flux, sample_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_frequency, frequency, sample_size, hipMemcpyHostToDevice );

    double one_over_2variance = 1 / (2 * variance);

    auto const num_blocks = ceil(num_frequencies/ static_cast< float >(blocksize));

    periodogram_frequency<<< num_blocks, blocksize >>>(dev_time, dev_flux, dev_frequency, dev_periodogram, one_over_2variance, sample_size, num_frequencies);

    hipMemcpy( periodogram, dev_periodogram, sample_size, hipMemcpyDeviceToHost );
}


__global__
void periodogram_frequency(float *flux, float *time, float *frequency, float *periodogram, float one_over_2variance, size_t sample_size, size_t num_frequencies){
    // Calculate frequency index
    int const w = threadIdx.x + blockIdx.x * blockDim.x;

    if ( w < num_frequencies) {    
        double cos_sum_squared = 0llu, cos_squared_sum = 0llu, sin_sum_squared = 0llu, sin_squared_sum = 0llu;
        // Not sure if we can do this, maybe we have to use hipMalloc and pass in?
        double time2[sample_size], sins[sample_size], coss[sample_size];
        double frequency_2_pi = 2 * M_PI * frequency[w];

        for(int i = 0; i < sample_size ; i++){
            time2[i] = frequency_2_pi * time[i];
        }
        for (int i = 0 ; i < sample_size ; i++){
            coss[i] = cos(time2[i]);
        }
        for (int i = 0 ; i < sample_size ; i++){
            sins[i] = sin(time2[i]);
        }
        for(int i = 0; i< sample_size;i++){
            cos_sum_squared += flux[i] * coss[i];
            cos_squared_sum += coss[i] * coss[i];
            sin_sum_squared += flux[i] * sins[i];
            sin_squared_sum += sins[i] * sins[i];
        }
        cos_sum_squared = cos_sum_squared * cos_sum_squared;
        sin_sum_squared = sin_sum_squared * sin_sum_squared;

        periodogram[w] = one_over_2variance * (cos_sum_squared/cos_squared_sum + sin_sum_squared/sin_squared_sum);
    }
}

