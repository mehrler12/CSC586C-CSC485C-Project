#include "hip/hip_runtime.h"

#include <iostream> // std::cout, std::endl
#include <chrono>   // std::chrono
#include <numeric>  // std::accumulate()
#include <algorithm>
#include <vector>
#include "../GPU_Data/cyclic-data.hpp"
// I changed this to .cu to get the correct highlighting
#include "lomb-scargle.cu"

int main()
{

    auto const benchmark_trials = 2000u;
    auto const threshold = 500llu;

    auto start_time = std::chrono::system_clock::now();

    double periodogram[sizeof(frequency)/sizeof(frequency[0])];
    for( auto i = 0u; i < benchmark_trials; ++i )
    {
        lomb_scargle(input_data,times,frequencies,periodogram,.1);
    }


    auto end_time = std::chrono::system_clock::now();
    auto elapsed_time = std::chrono::duration_cast< std::chrono::microseconds >( end_time - start_time );

    double max = 0;
    int index;
    for (int i = 0; i < sizeof(frequency)/sizeof(frequency[0]); i++ ){
        if(periodogram[i] > max) {
            max = periodogram[i];
            index = i;
        }
    }
    std::cout << "max = " << max << "\n";
    std::cout << "index = " << frequencies[index] << "\n";
    std::cout << "average time per run: "
              << elapsed_time.count() / static_cast< float >( benchmark_trials )
              << " us" << std::endl;

    return 0;
}